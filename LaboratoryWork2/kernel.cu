﻿#include <iostream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

#define N 1024
#define GRID_SIZE 128
#define BLOCK_SIZE 128

__global__ void PiCalcGPU(float* res, hiprandState* states) {
	unsigned long index = threadIdx.x + blockDim.x * blockIdx.x;
	int V = 0;
	float x, y;
	//curand для генерации случайных чисел на GPU
	hiprand_init(index, index, 0, &states[index]);

	for (int i = 0; i < N; i++) {
		//создаем последовательности значений x и y
		x = hiprand_uniform(&states[index]);
		y = hiprand_uniform(&states[index]);
		//рассчитываем V для значений
		V += (x * x + y * y <= 1.0f);
	}
	res[index] = 4.0f * V / (float)N;
}

float PiCalcGPU(long n) {
	float x, y;
	long V = 0;
	for (long i = 0; i < n; i++) {
		x = rand() / (float)RAND_MAX;
		y = rand() / (float)RAND_MAX;
		V += (x * x + y * y <= 1.0f);
	}
	return 4.0f * V / n;
}

int main(int argc, char* argv[]) {
	setlocale(LC_ALL, "Russian");
	//переменные времени
	clock_t start, stop;
	float host[GRID_SIZE * BLOCK_SIZE];
	float* device;
	hiprandState* hiprand;

	//Вычисление на GPU
	//Старт
	start = clock();
	//Выделение памяти
	hipError_t cuerr = hipMalloc((void**)&device, GRID_SIZE * BLOCK_SIZE * sizeof(float));
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate device: %s\n",
			hipGetErrorString(cuerr));
		return 0;
	}
	cuerr = hipMalloc((void**)&hiprand, BLOCK_SIZE * GRID_SIZE * sizeof(hiprandState));
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot allocate device: %s\n",
			hipGetErrorString(cuerr));
		return 0;
	}
	//Запуск ядра
	PiCalcGPU <<< GRID_SIZE, BLOCK_SIZE >>> (device, hiprand);
	cuerr = hipGetLastError();
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot launch CUDA kernel: %s\n",
			hipGetErrorString(cuerr));
		return 0;
	}
	//Копируем результат с девайса на хост
	cuerr = hipMemcpy(host, device, GRID_SIZE * BLOCK_SIZE * sizeof(float), hipMemcpyDeviceToHost);
	if (cuerr != hipSuccess)
	{
		fprintf(stderr, "Cannot copy from device to host: %s\n",
			hipGetErrorString(cuerr));
		return 0;
	}
	float PI_GPU = 0;
	for (int i = 0; i < GRID_SIZE * BLOCK_SIZE; i++) {
		PI_GPU += host[i];
	}
	PI_GPU /= (GRID_SIZE * BLOCK_SIZE);
	stop = clock();

	printf("GPU PI = %f\n", PI_GPU);
	printf("Время работы на GPU %f c\n", (stop - start) / (float)CLOCKS_PER_SEC);

	//Вычисление на CPU
	start = clock();
	float cpuPI = PiCalcGPU(GRID_SIZE * BLOCK_SIZE * N);
	stop = clock();
	printf("CPU PI = %f\n", cpuPI);
	printf("Время работы на СPU %f c.\n", (stop - start) / (float)CLOCKS_PER_SEC);

	return 0;
}
